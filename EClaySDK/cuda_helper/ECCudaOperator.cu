#ifdef __APPLE__
#define EC_ENV_OS_APPLE
#else
#endif


#include <hip/hip_runtime.h>
#include <cutil.h>
#include <cutil_inline.h>

#ifdef EC_ENV_OS_APPLE
#include <shrQATest.h>
#include <shrUtils.h>
#endif



extern "C" {
    
    void gpuInit( int argc, char** argv ) {
#ifdef EC_ENV_OS_APPLE
        shrQAStart(argc, argv);
#endif
        CUT_DEVICE_INIT(argc, argv);
    }
    
    void gpuExit( int argc, char** argv ) {
#ifdef EC_ENV_OS_APPLE
        shrEXIT(argc, (const char**)argv);
#endif
        CUT_EXIT(argc, argv);
    }
    
    
    
    void gpuMalloc( void*& dev_ptr, int size ) {
        CUDA_SAFE_CALL( hipMalloc( (void**)&dev_ptr, size ) );
    }
    
    void gpuFree( void*& dev_ptr ) {
        CUDA_SAFE_CALL( hipFree( dev_ptr ) );
        dev_ptr = 0;
    }
    
    void gpuMemcpyDeviceToHost( void*& host_ptr, void*& dev_ptr, int size ) {
        CUDA_SAFE_CALL( hipMemcpy( host_ptr, dev_ptr, size, hipMemcpyDeviceToHost ) );
    }
    
    void gpuMemcpyHostToDevice( void*& dev_ptr, void*& host_ptr, int size ) {
        CUDA_SAFE_CALL( hipMemcpy( dev_ptr, host_ptr, size, hipMemcpyHostToDevice ) );
    }
    
    void gpuMemcpyDeviceToDevice( void*& to, void*& from, int size ) {
        CUDA_SAFE_CALL( hipMemcpy( to, from, size, hipMemcpyDeviceToDevice ) );
    }
    
    void gpuBindTexture( const texture<float>& texture, const float* texElmArray, const int size ) {
        CUDA_SAFE_CALL( hipBindTexture( NULL, texture, texElmArray, size ) );
    }
    void gpuUnbindTexture( const texture<float>& texture ) {
        CUDA_SAFE_CALL( hipUnbindTexture( texture ) );
    }
    
}

